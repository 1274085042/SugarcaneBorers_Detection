/** @file vl_nnconv.cu
 ** @brief Convolution block
 ** @author Andrea Vedaldi
 **/

/*
 Copyright (C) 2014 Andrea Vedaldi and Max Jaderberg.
 All rights reserved.

 This file is part of the VLFeat library and is made available under
 the terms of the BSD license (see the COPYING file).
 */

#include "bits/mexutils.h"
#include "bits/nnhelper.h"
#include "bits/im2col.hpp"
#include "bits/subsample.hpp"

#include <assert.h>

#include <blas.h>
#ifdef ENABLE_GPU
#include <hipblas.h>
#endif

/* option codes */
enum {
  opt_stride = 0,
  opt_pad,
  opt_verbose,
  opt_no_der_data,
  opt_no_der_filters,
  opt_no_der_biases,
} ;

/* options */
vlmxOption  options [] = {
  {"Stride",           1,   opt_stride             },
  {"Pad",              1,   opt_pad                },
  {"Verbose",          0,   opt_verbose            },
  {"NoDerData",        0,   opt_no_der_data        },
  {"NoDerFilters",     0,   opt_no_der_filters     },
  {"NoDerBiases",      0,   opt_no_der_biases      },
  {0,                  0,   0                      }
} ;

/* ---------------------------------------------------------------- */
/*                                                            Cache */
/* ---------------------------------------------------------------- */

#ifdef ENABLE_GPU
bool cublasInitialized = false ;
hipblasHandle_t thisCublasHandle ;
#endif

bool persistentDataInitialized = false ;
PackedData temp ;
PackedData allOnes ;

void atExit()
{
  packed_data_deinit (&temp)  ;
  packed_data_deinit (&allOnes)  ;
#ifdef ENABLE_GPU
  if (cublasInitialized) {
    hipblasDestroy(thisCublasHandle) ;
    cublasInitialized = false ;
  }
#endif
}

/* ---------------------------------------------------------------- */
/*                                                  Dispatcher func */
/* ---------------------------------------------------------------- */

static void
sgemv_dispatch(bool gpuMode,
               char op,
               ptrdiff_t m, ptrdiff_t n,
               float alpha,
               float const * a, ptrdiff_t lda,
               float const * x, ptrdiff_t incx,
               float beta,
               float * y, ptrdiff_t incy)
{
  if (!gpuMode) {
    sgemv(&op,
          &m, &n, &alpha,
          (float*)a, &lda,
          (float*)x, &incx,
          &beta,
          y, &incy) ;
  } else {
#ifdef ENABLE_GPU
    hipblasSgemv(thisCublasHandle,
                (op == 't') ? HIPBLAS_OP_T : HIPBLAS_OP_N,
                (int)m, (int)n,
                &alpha,
                a, lda,
                x, (int)incx,
                &beta,
                y, (int)incy) ;
#endif
  }
}

static void
sgemm_dispatch(bool gpuMode,
               char op1, char op2,
               ptrdiff_t m, ptrdiff_t n, ptrdiff_t k,
               float alpha,
               float const * a, ptrdiff_t lda,
               float const * b, ptrdiff_t ldb,
               float beta,
               float * c, ptrdiff_t ldc)
{
  if (!gpuMode) {
    sgemm(&op1, &op2,
          &m, &n, &k,
          &alpha,
          (float*)a, &lda,
          (float*)b, &ldb,
          &beta,
          c, &ldc) ;
  } else {
#ifdef ENABLE_GPU
    hipblasSgemm(thisCublasHandle,
                (op1 == 't') ? HIPBLAS_OP_T : HIPBLAS_OP_N,
                (op2 == 't') ? HIPBLAS_OP_T : HIPBLAS_OP_N,
                (int)m, (int)n, (int)k,
                &alpha,
                a, (int)lda,
                b, (int)ldb,
                &beta,
                c, (int)ldc);
#endif
  }
}

static void
copy_dispatch(bool gpuMode,
              float * dest,
              float const * src,
              size_t numElements)
{
  if (!gpuMode) {
    memcpy(dest, src, numElements * sizeof(float)) ;
  } else {
#ifdef ENABLE_GPU
    hipMemcpy(dest, src, numElements * sizeof(float), hipMemcpyDeviceToDevice) ;
#endif
  }
}

static void
subsample_dispatch(bool gpuMode,
                   float* subsampled,
                   float const* data,
                   size_t width,
                   size_t height,
                   size_t depth,
                   size_t strideX,
                   size_t strideY,
                   size_t padLeft,
                   size_t padRight,
                   size_t padTop,
                   size_t padBottom)
{
  if (!gpuMode) {
    subsample_cpu(subsampled,
                  data,
                  width,
                  height,
                  depth,
                  strideX,
                  strideY,
                  padLeft,
                  padRight,
                  padTop,
                  padBottom) ;
  } else {
#ifdef ENABLE_GPU
    subsample_gpu(subsampled,
                  data,
                  width,
                  height,
                  depth,
                  strideX,
                  strideY,
                  padLeft,
                  padRight,
                  padTop,
                  padBottom) ;
#endif
  }
}

static void
subsampleBackward_dispatch(bool gpuMode,
                           float* dzdx,
                           float const* dzdy,
                           size_t width,
                           size_t height,
                           size_t depth,
                           size_t strideX,
                           size_t strideY,
                           size_t padLeft,
                           size_t padRight,
                           size_t padTop,
                           size_t padBottom)
{
  if (!gpuMode) {
    subsampleBackward_cpu(dzdx,
                          dzdy,
                          width,
                          height,
                          depth,
                          strideX,
                          strideY,
                          padLeft,
                          padRight,
                          padTop,
                          padBottom) ;
  } else {
#ifdef ENABLE_GPU
    subsampleBackward_gpu(dzdx,
                          dzdy,
                          width,
                          height,
                          depth,
                          strideX,
                          strideY,
                          padLeft,
                          padRight,
                          padTop,
                          padBottom) ;
#endif
  }
}


static void
im2col_dispatch(bool gpuMode,
                float* stacked,
                float const* data,
                size_t width,
                size_t height,
                size_t depth,
                size_t windowWidth,
                size_t windowHeight,
                size_t strideX,
                size_t strideY,
                size_t padLeft,
                size_t padRight,
                size_t padTop,
                size_t padBottom)
{
  if (!gpuMode) {
    im2col_cpu<float>(stacked,
                      data,
                      width,
                      height,
                      depth,
                      windowWidth,
                      windowHeight,
                      strideX,
                      strideY,
                      padLeft,
                      padRight,
                      padTop,
                      padBottom) ;
  } else {
#ifdef ENABLE_GPU
    im2col_gpu<float>(stacked,
                      data,
                      width,
                      height,
                      depth,
                      windowWidth,
                      windowHeight,
                      strideX,
                      strideY,
                      padLeft,
                      padRight,
                      padTop,
                      padBottom) ;
#endif
  }
}

static void
col2im_dispatch(bool gpuMode,
                float* data,
                float const* stacked,
                size_t width,
                size_t height,
                size_t depth,
                size_t windowWidth,
                size_t windowHeight,
                size_t strideX,
                size_t strideY,
                size_t padLeft,
                size_t padRight,
                size_t padTop,
                size_t padBottom)
{
  if (!gpuMode) {
    col2im_cpu<float>(data,
                      stacked,
                      width,
                      height,
                      depth,
                      windowWidth,
                      windowHeight,
                      strideX,
                      strideY,
                      padLeft,
                      padRight,
                      padTop,
                      padBottom) ;
  } else {
#ifdef ENABLE_GPU
    col2im_gpu<float>(data,
                      stacked,
                      width,
                      height,
                      depth,
                      windowWidth,
                      windowHeight,
                      strideX,
                      strideY,
                      padLeft,
                      padRight,
                      padTop,
                      padBottom) ;
#endif
  }
}

/* ---------------------------------------------------------------- */
/*                                                       MEX driver */
/* ---------------------------------------------------------------- */

enum {
  IN_DATA = 0, IN_FILTERS, IN_BIASES, IN_DEROUTPUT, IN_END
} ;

enum {
  OUT_RESULT = 0, OUT_DERFILTERS, OUT_DERBIASES, OUT_END
} ;

void mexFunction(int nout, mxArray *out[],
                 int nin, mxArray const *in[])
{
  /* inputs */
  PackedData data ;
  PackedData filters ;
  PackedData biases ;
  PackedData derOutput ;

  /* outputs */
  PackedData output ;
  PackedData derData  ;
  PackedData derFilters ;
  PackedData derBiases ;

  PackedDataGeometry outputGeom ;
  PackedDataGeometry derDataGeom  ;
  PackedDataGeometry derFiltersGeom ;
  PackedDataGeometry derBiasesGeom ;
  PackedDataGeometry tempGeom ;
  PackedDataGeometry allOnesGeom ;

  int strideX = 1 ;
  int strideY = 1 ;
  int padLeft = 0 ;
  int padRight = 0 ;
  int padTop = 0 ;
  int padBottom = 0 ;
  int numGroups = 1 ;

#if ENABLE_GPU
  hipblasStatus_t stat;
  bool gpuMode = false ;
#else
  bool const gpuMode = false ;
#endif
  bool backMode = false ;
  bool hasFilters = false ;
  bool hasBiases = false ;
  bool fullyConnectedMode = false ;
  bool computeDerData = true ;
  bool computeDerFilters = true ;
  bool computeDerBiases = true ;

  int verbosity = 0 ;
  int opt ;
  int next = IN_END ;
  mxArray const *optarg ;

  packed_data_init_empty(&data) ;
  packed_data_init_empty(&filters) ;
  packed_data_init_empty(&biases) ;
  packed_data_init_empty(&derOutput) ;
  packed_data_init_empty(&output) ;
  packed_data_init_empty(&derData) ;
  packed_data_init_empty(&derFilters) ;
  packed_data_init_empty(&derBiases) ;
  if (!persistentDataInitialized) {
    persistentDataInitialized = true ;
    packed_data_init_empty(&temp) ;
    packed_data_init_empty(&allOnes) ;
  }

  /* -------------------------------------------------------------- */
  /*                                            Check the arguments */
  /* -------------------------------------------------------------- */

  mexAtExit(atExit) ;

  if (nin < 3) {
    mexErrMsgTxt("There are less than three arguments.") ;
  }

  if (nin > 3 && vlmxIsString(in[3],-1)) {
    next = 3 ;
    backMode = 0 ;
  } else {
    backMode = (nin >= 4) ;
  }

  while ((opt = vlmxNextOption (in, nin, options, &next, &optarg)) >= 0) {
    switch (opt) {
      case opt_verbose :
        ++ verbosity ;
        break ;

      case opt_stride :
        if (!vlmxIsPlainMatrix(optarg,-1,-1)) {
          mexErrMsgTxt("STRIDE is not a plain matrix.") ;
        }
        switch (mxGetNumberOfElements(optarg)) {
          case 1:
            strideY = (int)mxGetPr(optarg)[0] ;
            strideX = strideY ;
            break ;
          case 2:
            strideY = (int)mxGetPr(optarg)[0] ;
            strideX = (int)mxGetPr(optarg)[1] ;
            break ;
          default:
            mexErrMsgTxt("STRIDE has neither one nor two elements.") ;
        }
        break ;

      case opt_pad :
        if (!vlmxIsPlainMatrix(optarg,-1,-1)) {
          mexErrMsgTxt("PAD is not a plain matrix.") ;
        }
        switch (mxGetNumberOfElements(optarg)) {
          case 1:
            padLeft = (int)mxGetPr(optarg)[0] ;
            padRight = padLeft ;
            padTop = padLeft ;
            padBottom = padLeft ;
            break ;
          case 4:
            padTop = (int)mxGetPr(optarg)[0] ;
            padBottom = (int)mxGetPr(optarg)[1] ;
            padLeft = (int)mxGetPr(optarg)[2] ;
            padRight = (int)mxGetPr(optarg)[3] ;
            break ;
          default:
            mexErrMsgTxt("STRIDE has neither one nor two elements.") ;
        }
        break ;

      case opt_no_der_data :
        computeDerData = VL_FALSE ;
        break ;

      case opt_no_der_filters :
        computeDerFilters = VL_FALSE ;
        break ;

      case opt_no_der_biases :
        computeDerBiases = VL_FALSE ;
        break ;

      default: break ;
    }
  }

  packed_data_init_with_array(&data, in[IN_DATA]) ;
  packed_data_init_with_array(&filters, in[IN_FILTERS]) ;
  packed_data_init_with_array(&biases, in[IN_BIASES]) ;
  if (backMode) { packed_data_init_with_array(&derOutput, in[IN_DEROUTPUT]) ; }

#if ENABLE_GPU
  gpuMode = (data.mode == matlabGpuArrayWrapper) ;
  if (gpuMode) {
    mxInitGPU() ;
    if (!cublasInitialized) {
      stat = hipblasCreate(&thisCublasHandle) ;
      if (stat != HIPBLAS_STATUS_SUCCESS) {
        mexErrMsgTxt("Could not initialize cuBLAS.") ;
      }
      cublasInitialized = true ;
    }
  }
#endif

  hasFilters = filters.geom.numElements > 0 ;
  hasBiases = biases.geom.numElements > 0 ;

  /* check for GPU/data class consistency */
  if (gpuMode && (filters.mode != matlabGpuArrayWrapper & hasFilters)) {
    mexErrMsgTxt("DATA is a GPU array but FILTERS is not.") ;
  }
  if (gpuMode && (biases.mode != matlabGpuArrayWrapper & hasBiases)) {
    mexErrMsgTxt("DATA is a GPU array but BIASES is not.") ;
  }
  if (gpuMode && (derOutput.mode != matlabGpuArrayWrapper & backMode)) {
    mexErrMsgTxt("DATA is a GPU array but DEROUTPUT is not.") ;
  }
  if (data.geom.classID != mxSINGLE_CLASS) {
    mexErrMsgTxt("DATA is not of class SINGLE.");
  }
  if (hasFilters && filters.geom.classID != mxSINGLE_CLASS) {
    mexErrMsgTxt("FILTERS is not of class SINGLE.");
  }
  if (hasBiases && (biases.geom.classID != mxSINGLE_CLASS)) {
    mexErrMsgTxt("BIASES is not of class SINGLE.");
  }
  if (backMode && (derOutput.geom.classID != mxSINGLE_CLASS)) {
    mexErrMsgTxt("DEROUTPUT is not of class SINGLE.");
  }

  if (strideX < 1 || strideY < 1) {
    mexErrMsgTxt("At least one element of STRIDE is smaller than one.") ;
  }

  if (!hasFilters) {
    /*
     Specifying empty filters assumes that they act as the
     identity matrix. Geometrically, emulate this as data.geom.detph
     fiilters of size 1x1xdata.geom.depth.
     */
    filters.geom.width = 1 ;
    filters.geom.height = 1 ;
    filters.geom.depth = data.geom.depth ;
    filters.geom.size = data.geom.depth ;
  }
  packed_data_geom_init(&outputGeom,
                        mxSINGLE_CLASS,
                        (data.geom.height + (padTop+padBottom) - filters.geom.height)/strideY + 1,
                        (data.geom.width + (padLeft+padRight) - filters.geom.width)/strideX + 1,
                        filters.geom.size,
                        data.geom.size) ;

  /* grouped filters */
  numGroups = data.geom.depth / filters.geom.depth ;

  /* if the output is 1x1 pixels, then there is no need to actually
   call im2col as it does not do anything
   */
  fullyConnectedMode = (outputGeom.height == 1 &&
                        outputGeom.width == 1 &&
                        padTop == 0 &&
                        padBottom == 0 &&
                        padLeft == 0 &&
                        padRight == 0 &&
                        numGroups == 1) ;

  derDataGeom = data.geom ;
  derFiltersGeom = filters.geom ;
  if (hasBiases) {
    if (fullyConnectedMode) {
      packed_data_geom_init (&allOnesGeom, mxSINGLE_CLASS,
                             1, 1,
                             1, data.geom.size) ;
    } else {
      packed_data_geom_init (&allOnesGeom, mxSINGLE_CLASS,
                             outputGeom.height,
                             outputGeom.width,
                             1, 1) ;
    }
    derBiasesGeom = biases.geom ;
  } else {
    packed_data_geom_init (&allOnesGeom, mxSINGLE_CLASS,
                           0, 0, 0, 0) ;
  }

  packed_data_geom_init
  (&tempGeom,
   mxSINGLE_CLASS,
   outputGeom.height,
   outputGeom.width,
   filters.geom.height*filters.geom.width*filters.geom.depth*numGroups,
   1) ;

  if (verbosity > 0) {
    mexPrintf("vl_nnconv: mode %s; %s\n", gpuMode?"gpu":"cpu", backMode?"backward":"forward") ;
    mexPrintf("vl_nnconv: stride: [%d %d], pad: [%d %d %d %d], numGroups: %d, has bias: %d, has filters: %d, fully connected: %d\n",
              strideY, strideX,
              padTop, padBottom, padLeft, padRight,
              numGroups, hasBiases, hasFilters, fullyConnectedMode) ;
    packed_data_geom_display(&data.geom, "vl_nnconv: data") ;
    if (hasFilters) { packed_data_geom_display(&filters.geom, "vl_nnconv: filters") ; }
    if (hasBiases) { packed_data_geom_display(&biases.geom, "vl_nnconv: biases") ; }
    if (backMode) {
      packed_data_geom_display(&derOutput.geom, "vl_nnconv: derOutput") ;
      packed_data_geom_display(&derDataGeom, "vl_nnconv: derData") ;
      if (hasFilters) { packed_data_geom_display(&derFiltersGeom, "vl_nnconv: derFilters") ; }
      if (hasBiases) { packed_data_geom_display(&derBiasesGeom, "vl_nnconv: derBiases") ; }
    } else {
      packed_data_geom_display(&outputGeom, "vl_nnconv: output") ;
    }
    packed_data_geom_display(&tempGeom, "vl_nnconv: temp") ;
    packed_data_geom_display(&temp.geom, "vl_nnconv: temp (cached)") ;
    packed_data_geom_display(&allOnesGeom, "vl_nnconv: allOnes") ;
    packed_data_geom_display(&allOnes.geom, "vl_nnconv: allOnes (cached)") ;
  }

  if (backMode) {
    if (derOutput.geom.height != tempGeom.height ||
        derOutput.geom.width != tempGeom.width ||
        derOutput.geom.depth != filters.geom.size ||
        derOutput.geom.size != data.geom.size)
    {
      mexErrMsgTxt("DEROUTPUT dimensions are incompatible with X and FILTERS.") ;
    }
  }

  if (numGroups * filters.geom.depth != data.geom.depth) {
    mexErrMsgTxt("The filter depth does not divide the image depth.") ;
  }

  if (filters.geom.size % numGroups != 0) {
    mexErrMsgTxt("The number of filter groups does not divide the total number of filters.") ;
  }

  if (padLeft < 0 ||
      padRight < 0 ||
      padTop < 0 ||
      padBottom < 0) {
    mexErrMsgTxt("An element of PAD is negative.") ;
  }

  if (data.geom.height + (padTop+padBottom) < filters.geom.height ||
      data.geom.width + (padLeft+padRight) < filters.geom.width) {
    mexErrMsgTxt("FILTERS are larger than the DATA (including padding).") ;
  }

  if (filters.geom.height == 0 || filters.geom.width == 0 || filters.geom.depth == 0) {
    mexErrMsgTxt("A dimension of FILTERS is void.") ;
  }

  if (hasBiases) {
    if (biases.geom.numElements != filters.geom.size) {
      mexErrMsgTxt("The number of elements of BIASES is not the same as the number of filters.") ;
    }
  }

  /* -------------------------------------------------------------- */
  /*                                                    Do the work */
  /* -------------------------------------------------------------- */

  /* auxiliary buffers */
  if (hasBiases) {
    if (allOnes.memorySize < allOnesGeom.numElements * sizeof(float) ||
        (allOnes.mode == matlabGpuArray || allOnes.mode == matlabGpuArrayWrapper) != gpuMode) {
      packed_data_deinit (&allOnes) ;
      packed_data_init_with_geom (&allOnes, gpuMode, allOnesGeom, true, true, 1.0f) ;
    }
  }
  if (!fullyConnectedMode) {
    if (temp.memorySize < tempGeom.numElements * sizeof(float) ||
        (temp.mode == matlabGpuArray || temp.mode == matlabGpuArrayWrapper) != gpuMode) {
      packed_data_deinit (&temp) ;
      packed_data_init_with_geom (&temp, gpuMode, tempGeom, true, false, 0);
    }
  }
  if (!backMode) {
    packed_data_init_with_geom(&output, gpuMode, outputGeom, false, false, 0) ;
  } else {
    if (computeDerData) {
      packed_data_init_with_geom(&derData, gpuMode, derDataGeom, false, false, 0) ;
    }
    if (computeDerFilters && hasFilters) {
      packed_data_init_with_geom(&derFilters, gpuMode, derFiltersGeom, false, false, 0) ;
    }
    if (computeDerBiases && hasBiases) {
      packed_data_init_with_geom(&derBiases, gpuMode, derBiasesGeom, false, false, 0) ;
    }
  }

  if (fullyConnectedMode) {
    float alpha = 1 ;
    float beta = 0 ;
    ptrdiff_t filtersVolume = filters.geom.height*filters.geom.width*filters.geom.depth ;
    /* note: fullyConnectedMode also guarantees no padding, num filter groups = 1 */

    /* optimise fully-connected mode case */
    if (!backMode) {
      if (hasFilters) {
        if (data.geom.size == 1) {
          /* one image in the stack */
          sgemv_dispatch(gpuMode, 't',
                         filtersVolume, filters.geom.size,
                         alpha,
                         filters.memory, filtersVolume,
                         data.memory, 1,
                         beta,
                         output.memory, 1) ;
        } else {
          /* multiple images in the stack */
          sgemm_dispatch(gpuMode, 't', 'n',
                         filters.geom.size, data.geom.size, filtersVolume,
                         alpha,
                         filters.memory, filtersVolume,
                         data.memory, filtersVolume,
                         beta,
                         output.memory, filters.geom.size) ;
        }
      } else {
        /* if no filter specified, assume that they act as the
         identity */
        copy_dispatch(gpuMode,
                      output.memory, data.memory,
                      filtersVolume * data.geom.size) ;
      }
      if (hasBiases) {
        float beta = 1 ;
        ptrdiff_t q = 1 ;
        sgemm_dispatch(gpuMode, 'n', 'n',
                       filters.geom.size, data.geom.size, q,
                       alpha,
                       biases.memory, filters.geom.size,
                       allOnes.memory, q,
                       beta,
                       output.memory, filters.geom.size) ;
      }
    } else {
      /* back mode */
      if (computeDerFilters && hasFilters) {
        sgemm_dispatch(gpuMode, 'n', 't',
                       filtersVolume, filters.geom.size, data.geom.size,
                       alpha,
                       data.memory, filtersVolume,
                       derOutput.memory, filters.geom.size,
                       beta,
                       derFilters.memory, filtersVolume) ;
      }
      if (computeDerBiases && hasBiases) {
        ptrdiff_t q = 1 ;
        sgemm_dispatch(gpuMode, 'n', 't',
                       q, filters.geom.size, data.geom.size,
                       alpha,
                       allOnes.memory, q,
                       derOutput.memory, filters.geom.size,
                       beta,
                       derBiases.memory, q) ;
      }
      if (computeDerData) {
        if (hasFilters) {
          sgemm_dispatch(gpuMode, 'n', 'n',
                         filtersVolume, data.geom.size, filters.geom.size,
                         alpha,
                         filters.memory, filtersVolume,
                         derOutput.memory, filters.geom.size,
                         beta,
                         derData.memory, filtersVolume) ;
        } else {
          /* does not have filters, just act as identity */
          copy_dispatch(gpuMode,
                        derData.memory, derOutput.memory,
                        filtersVolume * data.geom.size) ;
        }
      }
    }
  } else {
    /* not fully connected */
    for (int image = 0 ; image < data.geom.size ; ++image) {
      /*
       temp (phi(x)): m x k
       filters, derFilters: k x n (for one group of filters)
       derOutput (dzdy) : m x n (for one group of filters)
       res (y) : m x n (for one group of filters)
       */
      ptrdiff_t dataOffset = (data.geom.height*data.geom.width*data.geom.depth) * image ;
      ptrdiff_t outputOffset = (output.geom.height*output.geom.width*output.geom.depth) * image ;
      ptrdiff_t derDataOffset = (derData.geom.height*derData.geom.width*derData.geom.depth) * image ;
      ptrdiff_t derOutputOffset = (derOutput.geom.height*derOutput.geom.width*derOutput.geom.depth) * image ;
      ptrdiff_t m = tempGeom.height * tempGeom.width ; /* num output pixels */
      ptrdiff_t n = filters.geom.size/numGroups ; /* num filters per group */
      ptrdiff_t k = filters.geom.height*filters.geom.width*filters.geom.depth ; /* filter volume */

      if (backMode) {
        /* ---------------------------------------------------------- */
        /*                                              Backward mode */
        /* ---------------------------------------------------------- */

        /* compute derFilters dz/dF */
        if (computeDerFilters & hasFilters) {
          im2col_dispatch(gpuMode,
                          temp.memory,
                          data.memory + dataOffset,
                          data.geom.height, data.geom.width, data.geom.depth,
                          filters.geom.height, filters.geom.width,
                          strideY, strideX,
                          padTop, padBottom, padLeft, padRight) ;
          for (int g = 0 ; g < numGroups ; ++ g) {
            ptrdiff_t filterGrpOffset = k * n * g ;
            ptrdiff_t tempGrpOffset = m * k * g ;
            ptrdiff_t derOutputGrpOffset = m * n * g ;
            float alpha = 1 ;
            float beta = (image > 0) ; /* this saves init. the output array with 0 */
            sgemm_dispatch(gpuMode, 't', 'n',
                           k, n, m,
                           alpha,
                           (fullyConnectedMode ? data.memory : temp.memory)
                           + (fullyConnectedMode?dataOffset:0) + tempGrpOffset, m,
                           derOutput.memory + derOutputOffset + derOutputGrpOffset, m,
                           beta,
                           derFilters.memory + filterGrpOffset, k) ;
          }
        }

        /* compute derData dz/dbias */
        if (computeDerBiases & hasBiases) {
          sgemv_dispatch(gpuMode, 't',
                         m, filters.geom.size,
                         1, /* alpha */
                         derOutput.memory + derOutputOffset, m,
                         allOnes.memory, 1,
                         (float)(image > 0), /* beta */
                         derBiases.memory, 1) ;
        }

        /* compute derData dz/dx */
        if (computeDerData) {
          if (hasFilters) {
            for (int g = 0 ; g < numGroups ; ++ g) {
              ptrdiff_t filterGrpOffset = k * n * g ;
              ptrdiff_t tempGrpOffset = m * k * g ;
              ptrdiff_t derOutputGrpOffset = m * n * g ;
              float alpha = 1 ;
              float beta = fullyConnectedMode ? (g > 0) : 0 ;
              sgemm_dispatch(gpuMode, 'n', 't',
                             m, k, n,
                             alpha,
                             derOutput.memory + derOutputOffset + derOutputGrpOffset, m,
                             filters.memory + filterGrpOffset, k,
                             beta,
                             (fullyConnectedMode ? derData.memory : temp.memory)
                             + (fullyConnectedMode ? + derDataOffset : 0) + tempGrpOffset,
                             m) ;
            }
            col2im_dispatch(gpuMode,
                            derData.memory + derDataOffset,
                            temp.memory,
                            data.geom.height, data.geom.width, data.geom.depth,
                            filters.geom.height, filters.geom.width,
                            strideY, strideX,
                            padTop, padBottom, padLeft, padRight) ;
          } else {
            /* no filters: identity */
            subsampleBackward_dispatch(gpuMode,
                                       derData.memory + derDataOffset,
                                       derOutput.memory + derOutputOffset,
                                       data.geom.height, data.geom.width, data.geom.depth,
                                       strideY, strideX,
                                       padTop, padBottom, padLeft, padRight) ;
          }
        }
      } else {
        /* ---------------------------------------------------------- */
        /*                                               Forward mode */
        /* ---------------------------------------------------------- */
        if (hasFilters) {
          im2col_dispatch(gpuMode,
                          temp.memory,
                          data.memory + dataOffset,
                          data.geom.height, data.geom.width, data.geom.depth,
                          filters.geom.height, filters.geom.width,
                          strideY, strideX,
                          padTop, padBottom, padLeft, padRight) ;
          for (int g = 0 ; g < numGroups ; ++ g) {
            ptrdiff_t filterGrpOffset = k * n * g ;
            ptrdiff_t tempGrpOffset = m * k * g ;
            ptrdiff_t outputGrpOffset = m * n * g  ;
            float alpha = 1 ;
            float beta = 0 ;
            sgemm_dispatch(gpuMode, 'n', 'n',
                           m, n, k,
                           alpha,
                           (fullyConnectedMode ? data.memory : temp.memory)
                           + (fullyConnectedMode?dataOffset:0) + tempGrpOffset, m,
                           filters.memory + filterGrpOffset, k,
                           beta,
                           output.memory + outputOffset + outputGrpOffset, m) ;
          }
        } else {
          /* no filters: identity */
          subsample_dispatch(gpuMode,
                             output.memory + outputOffset,
                             data.memory + dataOffset,
                             data.geom.height, data.geom.width, data.geom.depth,
                             strideY, strideX,
                             padTop, padBottom, padLeft, padRight) ;
        }
        if (hasBiases) {
          float alpha = 1 ;
          float beta = 1 ;
          ptrdiff_t q = 1 ;
          sgemm_dispatch(gpuMode, 'n', 'n',
                         m, biases.geom.numElements, q,
                         alpha,
                         allOnes.memory, m,
                         biases.memory, q,
                         beta,
                         output.memory + outputOffset, m) ;
        }
      }
    }
  }

  /* -------------------------------------------------------------- */
  /*                                                        Cleanup */
  /* -------------------------------------------------------------- */

  packed_data_deinit(&data) ;
  packed_data_deinit(&filters) ;
  packed_data_deinit(&biases) ;
  if (backMode) {
    packed_data_deinit(&derOutput) ;
    out[OUT_RESULT] = (computeDerData) ? packed_data_deinit_extracting_array(&derData) : mxCreateDoubleMatrix(0,0,mxREAL) ;
    out[OUT_DERFILTERS] =(computeDerFilters & hasFilters)? packed_data_deinit_extracting_array(&derFilters) : mxCreateDoubleMatrix(0,0,mxREAL) ;
    out[OUT_DERBIASES] = (computeDerBiases & hasBiases) ? packed_data_deinit_extracting_array(&derBiases) : mxCreateDoubleMatrix(0,0,mxREAL) ;
  } else {
    out[OUT_RESULT] = packed_data_deinit_extracting_array(&output) ;
  }
}
