#include "hip/hip_runtime.h"
/** @file pooling_gpu.cu
 ** @brief Max pooling filters (GPU)
 ** @author Andrea Vedaldi
 ** @author Karel Lenc
 **/

/*
Copyright (C) 2014 Andrea Vedaldi and Karel Lenc.
All rights reserved.

This file is part of the VLFeat library and is made available under
the terms of the BSD license (see the COPYING file).
*/

#include "gpu.hpp"
#include "pooling.hpp"

#include <assert.h>
#include <float.h>
#include <sm_20_atomic_functions.h>

/* ---------------------------------------------------------------- */
/*                                                 maxPooling (GPU) */
/* ---------------------------------------------------------------- */

template<typename T>
__global__ void maxPooling_gpu_kernel
(T* pooled,
 const T* data,
 const int pooledWidth,
 const int pooledHeight,
 const int pooledVolume,
 const int width,
 const int height,
 const int windowWidth,
 const int windowHeight,
 const int strideX,
 const int strideY,
 const int padLeft,
 const int padTop)
{
  int pooledIndex = threadIdx.x + blockIdx.x * blockDim.x;
  if (pooledIndex < pooledVolume) {
    /* pooledIndex = x
                   + y * pooledWidth
                   + z * (pooledWidth * pooledHeight) ;
     */
    int px = pooledIndex ;
    int py = px / pooledWidth ;
    int pz = py / pooledHeight ;
    px %= pooledWidth ;
    py %= pooledHeight ;
    int x1 = px * strideX - padLeft ;
    int y1 = py * strideY - padTop ;
    int x2 = min(x1 + windowWidth, width) ;
    int y2 = min(y1 + windowHeight, height) ;
    x1 = max(x1, 0) ;
    y1 = max(y1, 0) ;
    data += pz * (width*height) ;
    T bestValue = data[y1 * width + x1] ;
    for (int y = y1 ; y < y2 ; ++y) {
      for (int x = x1 ; x < x2 ; ++x) {
        bestValue = max(bestValue, data[y * width + x]) ;
      }
    }
    pooled[pooledIndex] = bestValue ;
  }
}


template<typename T>
__global__ void avgPooling_gpu_kernel
(T* pooled,
 const T* data,
 const int pooledWidth,
 const int pooledHeight,
 const int pooledVolume,
 const int width,
 const int height,
 const int windowWidth,
 const int windowHeight,
 const int strideX,
 const int strideY,
 const int padLeft,
 const int padTop)
{
  int pooledIndex = threadIdx.x + blockIdx.x * blockDim.x;
  if (pooledIndex < pooledVolume) {
    /* pooledIndex = x
                   + y * pooledWidth
                   + z * (pooledWidth * pooledHeight) ;
     */
    int px = pooledIndex ;
    int py = px / pooledWidth ;
    int pz = py / pooledHeight ;
    px %= pooledWidth ;
    py %= pooledHeight ;
    int x1 = px * strideX - padLeft ;
    int y1 = py * strideY - padTop ;
    int x2 = min(x1 + windowWidth, width) ;
    int y2 = min(y1 + windowHeight, height) ;
    x1 = max(x1, 0) ;
    y1 = max(y1, 0) ;
    data += pz * (width*height) ;
    T accum = 0;
    T poolSize = (y2 - y1)*(x2 - x1);
    for (int y = y1 ; y < y2 ; ++y) {
      for (int x = x1 ; x < x2 ; ++x) {
        accum += data[y * width + x] ;
      }
    }
    pooled[pooledIndex] = accum / poolSize ;
  }
}


template<typename T>
void pooling_gpu(T* pooled,
                 T const* data,
                 PoolMethod method,
                 size_t width,
                 size_t height,
                 size_t depth,
                 size_t windowWidth,
                 size_t windowHeight,
                 size_t strideX,
                 size_t strideY,
                 size_t padLeft,
                 size_t padRight,
                 size_t padTop,
                 size_t padBottom)
{
  int pooledWidth = (width + (padLeft+padRight) - windowWidth)/strideX + 1 ;
  int pooledHeight = (height + (padTop+padBottom) - windowHeight)/strideY + 1 ;
  int pooledVolume = pooledWidth * pooledHeight * depth ;
  switch (method) {
    case NN_POOL_MAX :
      maxPooling_gpu_kernel<T>
      <<< divideUpwards(pooledVolume, VL_CUDA_NUM_THREADS), VL_CUDA_NUM_THREADS >>>
      (pooled, data,
       pooledWidth, pooledHeight, pooledVolume,
       width, height,
       windowWidth, windowHeight,
       strideX, strideY,
       padLeft, padTop);
      if (hipGetLastError() != hipSuccess) {
        std::cout
        <<"maxPooling_gpu_kernel error ("
        <<hipGetErrorString(hipGetLastError())
        <<")"<<std::endl ;
      }
      break;
  case NN_POOL_AVG :
    avgPooling_gpu_kernel<T>
    <<< divideUpwards(pooledVolume, VL_CUDA_NUM_THREADS), VL_CUDA_NUM_THREADS >>>
    (pooled, data,
     pooledWidth, pooledHeight, pooledVolume,
     width, height,
     windowWidth, windowHeight,
     strideX, strideY,
     padLeft, padTop);
    if (hipGetLastError() != hipSuccess) {
      std::cout
      <<"avgPooling_gpu_kernel error ("
      <<hipGetErrorString(hipGetLastError())
      <<")"<<std::endl ;
    }
    break;
  default:
    assert(false);
  }
}

template
void pooling_gpu<float>(float* pooled,
                        float const* data,
                        PoolMethod method,
                        size_t width,
                        size_t height,
                        size_t depth,
                        size_t windowWidth,
                        size_t windowHeight,
                        size_t strideX,
                        size_t strideY,
                        size_t padLeft,
                        size_t padRight,
                        size_t padTop,
                        size_t padBottom) ;

template
void pooling_gpu<double>(double* pooled,
                         double const* data,
                         PoolMethod method,
                         size_t width,
                         size_t height,
                         size_t depth,
                         size_t windowWidth,
                         size_t windowHeight,
                         size_t strideX,
                         size_t strideY,
                         size_t padLeft,
                         size_t padRight,
                         size_t padTop,
                         size_t padBottom) ;

/* ---------------------------------------------------------------- */
/*                                         maxPoolingBackward (GPU) */
/* ---------------------------------------------------------------- */

#ifdef VLNN_CAFFELIKE_BPPOOL
// In order to be able to use this, BP would need to have access to both
// bottom data and pooled data (currently only passed bottom data...)
template <typename T>
__global__ void maxPoolingBackward_gpu_kernel_caffelike(
    T* dzdx,
    const T* data,
    const T* pooled,
    const T* dzdy,
    const int nthreads,
    const int pooledWidth,
    const int pooledHeight,
    const int width,
    const int height,
    const int depth,
    const int windowWidth,
    const int windowHeight,
    const int strideX,
    const int strideY)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < nthreads) {
    // find out the local index
    // find out the local offset
    int x = index % width;
    int y = (index / width) % height;
    int z = (index / width / height) % depth;
    int py1 = (y < windowHeight) ? 0 : (y - windowHeight) / strideY + 1;
    int py2 = min(y / strideY + 1, pooledHeight);
    int px1 = (x < windowWidth) ? 0 : (x - windowWidth) / strideX + 1;
    int px2 = min(x / strideX + 1, pooledWidth);
    T gradient = 0;
    T datum = data[(z * height + y) * width + x];
    pooled += z * pooledHeight * pooledWidth;
    dzdy += z * pooledHeight * pooledWidth;
    for (int py = py1; py < py2; ++py) {
      for (int px = px1; px < px2; ++px) {
        gradient += dzdy[py * pooledWidth + px] *
            (datum == pooled[py * pooledWidth + px]);
      }
    }
    dzdx[index] = gradient;
  }
}
#endif


template<typename T>
__global__ void maxPoolingBackward_gpu_kernel
(T* dzdx,
 const T* data,
 const T* dzdy,
 const int pooledWidth,
 const int pooledHeight,
 const int pooledVolume,
 const int width,
 const int height,
 const int windowWidth,
 const int windowHeight,
 const int strideX,
 const int strideY,
 const int padLeft,
 const int padTop)
{
  int pooledIndex = threadIdx.x + blockIdx.x * blockDim.x;
  if (pooledIndex < pooledVolume) {
    /* pooledIndex = x
     + y * pooledWidth
     + z * (pooledWidth * pooledHeight) ;
     */
    int px = pooledIndex ;
    int py = px / pooledWidth ;
    int pz = py / pooledHeight ;
    px %= pooledWidth ;
    py %= pooledHeight ;
    data += pz * (width*height) ;
    dzdx += pz * (width*height) ;

    int x1 = px * strideX - padLeft ;
    int y1 = py * strideY - padTop ;
    int x2 = min(x1 + windowWidth, width) ;
    int y2 = min(y1 + windowHeight, height) ;
    x1 = max(x1, 0) ;
    y1 = max(y1, 0) ;
    int bestIndex = y1 * width + x1 ;
    T bestValue = data[bestIndex] ;
    for (int y = y1 ; y < y2 ; ++y) {
      for (int x = x1 ; x < x2 ; ++x) {
        int index = y * width + x ;
        T value = data[index] ;
        if (value > bestValue) {
          bestValue = value ;
          bestIndex = index ;
        }
      }
    }
    /*
     This is bad, but required to eliminate a race condition when writing
     to bottom_diff.
     Caffe goes the other way around, but requrires remembering the layer
     output, or the maximal indexes.
     atomicAdd(add, val)
     */
    atomicAdd(dzdx + bestIndex, dzdy[pooledIndex]) ;
  }
}

template <typename T>
__global__ void avgPoolingBackward_gpu_kernel(
    T* dzdx,
    const T* dzdy,
    const int nthreads,
    const int pooledWidth,
    const int pooledHeight,
    const int width,
    const int height,
    const int depth,
    const int windowWidth,
    const int windowHeight,
    const int strideX,
    const int strideY,
    const int padLeft,
    const int padTop)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < nthreads) {
    /* To understand the logic of this piece of code see the
     comments to col2im_gpu_kernel */
    int x_data = index ;
    int y_data = x_data / width ;
    int z = y_data / height ;
    x_data %= width ;
    y_data %= height ;

    int dx = x_data + padLeft - windowWidth ;
    int dy = y_data + padTop - windowHeight ;
    int px1 = (dx >= 0) ? dx/strideX + 1 : 0 ;
    int py1 = (dy >= 0) ? dy/strideY + 1 : 0 ;
    int px2 = min((x_data + padLeft) / strideX, pooledWidth - 1) ;
    int py2 = min((y_data + padTop) / strideY, pooledHeight - 1) ;
    T accumulator = 0 ;
    dzdy += z * pooledHeight * pooledWidth;
    for (int py = py1 ; py <= py2 ; ++py) {
      for (int px = px1 ; px <= px2 ; ++px) {
        int x1 = px * strideX - padLeft ;
        int y1 = py * strideY - padTop ;
        int x2 = min(x1 + windowWidth, width) ;
        int y2 = min(y1 + windowHeight, height) ;
        x1 = max(x1, 0) ;
        y1 = max(y1, 0) ;
        T poolSize = (y2 - y1) * (x2 - x1);
        accumulator += dzdy[py * pooledWidth + px] / poolSize ;
      }
    }
    dzdx[index] = accumulator ;
  }
}


template<typename T>
void poolingBackward_gpu(T* dzdx,
                         T const* data,
                         T const* dzdy,
                         PoolMethod method,
                         size_t width,
                         size_t height,
                         size_t depth,
                         size_t windowWidth,
                         size_t windowHeight,
                         size_t strideX,
                         size_t strideY,
                         size_t padLeft,
                         size_t padRight,
                         size_t padTop,
                         size_t padBottom)
{
  int pooledWidth = (width + (padLeft+padRight) - windowWidth)/strideX + 1 ;
  int pooledHeight = (height + (padTop+padBottom) - windowHeight)/strideY + 1 ;
  int nthreads;
  switch (method) {
    case NN_POOL_MAX:
      nthreads = pooledWidth * pooledHeight * depth ;
      maxPoolingBackward_gpu_kernel<T>
      <<< divideUpwards(nthreads, VL_CUDA_NUM_THREADS), VL_CUDA_NUM_THREADS >>>
      (dzdx,
       data, dzdy,
       pooledWidth, pooledHeight, nthreads,
       width, height,
       windowWidth, windowHeight,
       strideX, strideY,
       padLeft, padTop);
      if (hipGetLastError() != hipSuccess) {
        std::cout
        <<"maxPooling_gpu_kernel error ("
        <<hipGetErrorString(hipGetLastError())
        <<")"<<std::endl ;
      }
      break;
    case NN_POOL_AVG:
      nthreads = width * height * depth ;
      avgPoolingBackward_gpu_kernel<T>
      <<< divideUpwards(nthreads, VL_CUDA_NUM_THREADS), VL_CUDA_NUM_THREADS >>>
      (dzdx,
       dzdy,
       nthreads,
       pooledWidth,
       pooledHeight,
       width,
       height,
       depth,
       windowWidth,
       windowHeight,
       strideX,
       strideY,
       padLeft,
       padTop);
      if (hipGetLastError() != hipSuccess) {
        std::cout
        <<"avgPooling_gpu_kernel error ("
        <<hipGetErrorString(hipGetLastError())
        <<")"<<std::endl ;
      }
      break;
    default:
      assert(false) ;
  }
}

template
void poolingBackward_gpu<float>(float* dzdx,
                                float const* data,
                                float const* dzdy,
                                PoolMethod method,
                                size_t width,
                                size_t height,
                                size_t depth,
                                size_t windowWidth,
                                size_t windowHeight,
                                size_t strideX,
                                size_t strideY,
                                size_t padLeft,
                                size_t padRight,
                                size_t padTop,
                                size_t padBottom) ;

#if 0
template
void maxPoolingBackward_gpu<double>(double* dzdx,
                                    double const* data,
                                    double const* dzdy,
                                    size_t width,
                                    size_t height,
                                    size_t depth,
                                    size_t windowWidth,
                                    size_t windowHeight,
                                    size_t strideX,
                                    size_t strideY,
                                    size_t padLeft,
                                    size_t padRight,
                                    size_t padTop,
                                    size_t padBottom) ;
#endif
